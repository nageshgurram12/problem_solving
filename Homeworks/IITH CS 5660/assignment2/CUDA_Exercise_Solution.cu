//Submission should be named as  <RollNumber>_Prog.cu
//Upload just this cu file and nothing else. If you upload it as a zip, it will not be evaluated.



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define M 514
//Input has 514 rows and columns

#define N 512
//For output, only 512 rows and columns need to be computed.

#define BLOCKS 9
#define THREADSPERBLOCK 48

//TODO: WRITE GPU KERNEL. It should not be called repeatedly from the host, but just once. Each time it is called, it may process more than pixel or not process any pixel at all.
__global__ void image_blur(int* d_A, int* d_B, int pixelsPerThread){
  // Access all pixels for the given thread and calculate blur value for it
  int index = 0;
  int row = 0;
  int  column = 0;
  int i;
  for(i=0; i<pixelsPerThread; i++){
   index = i + (pixelsPerThread * threadIdx.x) + (blockIdx.x * blockDim.x) * pixelsPerThread;
    row = index % M;
    column = index / M;
     if(row > 0 && row < M-1 && column > 0 && column < M-1){
     d_B[row*M + column] = (d_A[(row-1)*M + column] + d_A[(row+1)*M + column] + d_A[row*M + (column-1)] + d_A[row*M + (column+1)])/4;
    }
  }
}

int main (int argc, char **argv) {
  int A[M][M], B[M][M];
  int *d_A, *d_B; // These are the copies of A and B on the GPU
  int *h_B;       // This is a host copy of the output of B from the GPU
  int i, j;

  // Input is randomly generated
  for(i=0;i<M;i++) {
    for(j=0;j<M;j++) {
      A[i][j] = rand()/1795831;
    }
  }
  // sequential implementation of main computation
  for(i=1;i<M-1;i++) {
    for(j=1;j<M-1;j++) {
      B[i][j] = (A[i-1][j]+A[i+1][j]+A[i][j-1]+A[i][j+1])/4;
    }
  }


  // TODO: ALLOCATE MEMORY FOR GPU COPIES OF d_A AND d_B
  int size = sizeof(int) *  M * M;

  hipMalloc((void **) & d_A, size);
  hipMalloc((void **) & d_B, size);

  h_B = (int *) malloc(size);

  // TODO: COPY A TO d_A
  hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);

  // TODO: CREATE BLOCKS with THREADS AND INVOKE GPU KERNEL
   //Use 9 blocks, each with 48 threads

  int pixelsPerThread = ceil((M * M) / (BLOCKS * THREADSPERBLOCK) + 0.0);
  image_blur<<<BLOCKS, THREADSPERBLOCK>>>(d_A, d_B, pixelsPerThread);
  hipDeviceSynchronize();


  // TODO: COPY d_B BACK FROM GPU to CPU in variable h_B
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);

  // TODO: Verify result is correct by comparing
  for(i=1;i<M-1;i++) {
    for(j=1;j<M-1;j++) {
    //TODO: compare each element of h_B and B by subtracting them
        //print only those elements for which the above subtraction is non-zero
      if(h_B[i*M+j] != B[i][j]){
       printf("!!!! Error at %d row and %d column for host value %d and device value %d !!!\n", i , j, B[i][j], h_B[i*M+j]);
      }
    }
   }

   hipFree(d_A);
   hipFree(d_B);
   free(h_B);
    //IF even one element of h_B and B differ, report an error.
    //Otherwise, there is no error.
    //If your program is correct, no error should occur.
}

/*Remember the following guidelines to avoid losing marks
Index of an array should not exceed the array size.
Do not ignore the fact that boundary rows and columns need not be computed (in fact, they cannot be computed since they don't have four neighbors)
No output array-element should be computed more than once
No marks will be given if the program does not compile or run (TAs will not debug your program at all)
*/

